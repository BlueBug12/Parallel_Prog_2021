#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int count){
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
          break;
        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int* output_device, int resX, int resY, int maxIterations, size_t pitch, int shift_x) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;
    if(j>=resY)
        return;
    float y = lowerY + j*stepY;
    
    for(int k=0;k<shift_x;++k){
        int s_i = i*shift_x+k;
        if(s_i>=resX)
            return;
        float x = lowerX + s_i*stepX;
        int index = j*pitch + s_i;
        output_device[index] = mandel(x,y,maxIterations);
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int *output_host; 
    int *output_device;
    size_t pitch;
    
    hipMallocPitch(&output_device,&pitch,sizeof(int)*resX,resY);
    hipHostAlloc(&output_host,resY*pitch,hipHostMallocMapped);
    
    int shift_x = 2;
    float block_dim_x = 16;
    float block_dim_y = 16;

    dim3 threadsPerBlock(block_dim_x,block_dim_y);
    dim3 numBlocks((int)(ceil(resX/block_dim_x/shift_x)),(int)ceil(resY/block_dim_y));
    mandelKernel<<<numBlocks,threadsPerBlock>>>(stepX,stepY,lowerX,lowerY,output_device,resX,resY,maxIterations,pitch/(sizeof(int)),shift_x);

    hipMemcpy(output_host,output_device,pitch*resY,hipMemcpyDeviceToHost);
    pitch/=sizeof(int);
    
    for(int i=0;i<resY;++i){
        for(int j=0;j<resX;++j){
            img[i*resX+j] = output_host[i*pitch+j];
        }
    }
    hipHostFree(output_host);
    hipFree(output_device);
}
