#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int count){
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
          break;
        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int* output_device, int resX, int resY, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;
    if(i>=resX||j>=resY)
        return;
    float x = lowerX + i*stepX;
    float y = lowerY + j*stepY;
    int index = j*resX + i;
    output_device[index] = mandel(x,y,maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int size = resX*resY*sizeof(int);
    int *output_host = (int *)malloc(size);
    int *output_device;
    hipMalloc(&output_device,size);
    float block_dim_x = 16;
    float block_dim_y = 16;
    dim3 threadsPerBlock(block_dim_x,block_dim_y);
    dim3 numBlocks((int)ceil(resX/block_dim_x),(int)ceil(resY/block_dim_y));
    mandelKernel<<<numBlocks,threadsPerBlock>>>(stepX,stepY,lowerX,lowerY,output_device,resX,resY,maxIterations);

    hipMemcpy(output_host,output_device,size,hipMemcpyDeviceToHost);
    memcpy(img,output_host,size);
    free(output_host);
    hipFree(output_device);
}
